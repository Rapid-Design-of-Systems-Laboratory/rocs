#include "hip/hip_runtime.h"
#include <stdio.h>
#define _num_t %DATATYPE%
#define EPS 1e-8

__device__ __forceinline__ int compare(double a, double b, double tol)
{
	if(a > b && abs(a-b) > tol)
		return 1;
	else if(a < b && abs(a-b) > tol)
		return -1;
	else
		return 0;
}
__device__ __forceinline__ bool isNaN(double a)
{
	return isnan(a);
}

#include "computeControl.h"
__device__ double calc_xdot(const int _i, const int _arcId, const int _arcType, double *d_data, double *d_const, double *d_constraints, const int N)
{
%STATE_FUNCTIONS%
}

/****************************************************************************

Optimized version of STM propagator using multiple sub-arcs and multiple arcs

*****************************************************************************/

extern "C" __global__ 
	void stt1_Fstar_kernel(double *d_x0, double *d_F, double *d_const, double *d_constraints, 
		double* d_phi0, double *d_k, double *d_k2, double *d_phi1, const double h,
		const double k_mul1, const double k_mul2, const int N, const int P, double *arcSequence, int x0Size, int part)
{
	double x0_[%NUMSTATES%+%MAXARCS%];
	// complex_t x0_[N];
	int j;
	int realcol, realrow;

	if(part == 1)
	{
		// Copy states for every sub-arc into shared memory for current sub-step
		// N values per thread, P threads
		if(blockIdx.x<N && blockIdx.y<N && threadIdx.x < P)
		{
			
			for(j=0;j<N;j++)
			{
				realcol = blockIdx.z*x0Size + j*P+ threadIdx.x;
				x0_[j] = d_x0[realcol];
			}			
			realcol = P*blockIdx.y + threadIdx.x;
			
			double fx = calc_xdot(blockIdx.x,blockIdx.z, arcSequence[blockIdx.z], x0_, d_const, d_constraints, N);
			x0_[blockIdx.y] += EPS;
			fx = calc_xdot(blockIdx.x,blockIdx.z, arcSequence[blockIdx.z], x0_, d_const, d_constraints, N)/EPS - fx/EPS;
			d_F[blockIdx.z*N*N*P + N*P*blockIdx.x + realcol] = fx;
			// if(blockIdx.z == 1 && blockIdx.y == 14 && blockIdx.x == 6)
			// {
			// 	double bank, alfa,hamiltonian;
			// 	computeControlConstraint1(d_x0,d_const,d_constraints,&bank,&alfa,&hamiltonian);
			// 	printf("%%d, %%d, %%lf, alfa = %%lf, bank = %%lf\\n",blockIdx.x, blockIdx.y, fx/EPS, alfa, bank);
			// }
			
		}
	}
	else if(part == 2)
	{
		// Part 2
		double phidot;
		if(blockIdx.x<N && blockIdx.y<N && threadIdx.x < P)
		{
			// Dot product of F and phi
			phidot = 0;
			for(j=0;j<N;j++)
			{
				// realrow = j*P + threadIdx.x;
				realcol = j*P + threadIdx.x;
				phidot += d_F[blockIdx.z*N*N*P + N*P*blockIdx.x + realcol]*(d_phi0[blockIdx.z*N*N*P + blockIdx.y*N*P + realcol] + d_k[blockIdx.z*N*N*P + blockIdx.y*N*P + realcol]*k_mul2);
			}
			phidot = h*phidot;
			// Store into "k" matrix in col-major form  k_i = h*phidot
			realrow = blockIdx.x*P + threadIdx.x;
			
			d_k2  [blockIdx.z*N*N*P + blockIdx.y*N*P + realrow] = phidot;
			d_phi1[blockIdx.z*N*N*P + blockIdx.y*N*P + realrow] += k_mul1*phidot;		
		}
	}
}

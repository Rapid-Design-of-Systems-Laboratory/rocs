#include "hip/hip_runtime.h"
// 
#include <stdio.h>
#include "cusp/complex.h"
#define EPS 1e-50
#define complex_t cusp::complex<double>
#define _num_t %DATATYPE%

__device__ complex_t fabs(complex_t a)
{
	a.real(fabs(a.real()));
	return a;
}
__device__ bool isNaN(complex_t a)
{
	return isnan(a.real());
}
__device__ int compare(complex_t a, complex_t b, double tol)
{
	if(norm(a) > norm(b) && abs(norm(a)-norm(b)) > tol)
		return 1;
	else if(norm(a) < norm(b) && abs(norm(a)-norm(b)) > tol)
		return -1;
	else
		return 0;
}
#include "computeControl.h"

__device__ complex_t calc_xdot(const int _i, const int _arcId, const int _arcType, complex_t *d_data, double *d_const, double *d_constraints, const int N)
{
%STATE_FUNCTIONS%
}

/****************************************************************************

Optimized version of STM propagator using multiple sub-arcs and multiple arcs

*****************************************************************************/

extern "C" __global__ 
	void stt1_Fstar_kernel(double *d_x0, double *d_F, double *d_const, double *d_constraints, 
		double* d_phi0, double *d_k, double *d_k2, double *d_phi1, const double h,
		const double k_mul1, const double k_mul2, const int N, const int P, double *arcSequence, int x0Size, int part)
{
	complex_t x0_[%NUMSTATES%+%MAXARCS%];
	int j;
	int realcol, realrow;

	if(part == 1)
	{
		// Copy states for every sub-arc into shared memory for current sub-step
		// N values per thread, P threads
		if(blockIdx.x<N && blockIdx.y<N && threadIdx.x < P)
		{
			
			for(j=0;j<N;j++)
			{
				realcol = blockIdx.z*x0Size + j*P+ threadIdx.x;
				x0_[j].real(d_x0[realcol]);
				x0_[j].imag(0);
			}
			x0_[blockIdx.y].imag(EPS);	// Variable w.r.t which derivative is to be computed
			
			realcol = P*blockIdx.y + threadIdx.x;
			
			complex_t fx = calc_xdot(blockIdx.x, blockIdx.z, arcSequence[blockIdx.z], x0_, d_const, d_constraints, N);
			d_F[blockIdx.z*N*N*P + N*P*blockIdx.x + realcol] = fx.imag()/EPS;
		}
	}
	else if(part == 2)
	{
		// Part 2
		double phidot;
		if(blockIdx.x<N && blockIdx.y<N && threadIdx.x < P)
		{
			// Dot product of F and phi
			phidot = 0;
			for(j=0;j<N;j++)
			{
				// realrow = j*P + threadIdx.x;
				realcol = j*P + threadIdx.x;
				phidot += d_F[blockIdx.z*N*N*P + N*P*blockIdx.x + realcol]*(d_phi0[blockIdx.z*N*N*P + blockIdx.y*N*P + realcol] + d_k[blockIdx.z*N*N*P + blockIdx.y*N*P + realcol]*k_mul2);
			}
			phidot = h*phidot;
			// Store into "k" matrix in col-major form  k_i = h*phidot
			realrow = blockIdx.x*P + threadIdx.x;
			
			d_k2  [blockIdx.z*N*N*P + blockIdx.y*N*P + realrow] = phidot;
			d_phi1[blockIdx.z*N*N*P + blockIdx.y*N*P + realrow] += k_mul1*phidot;		
		}
	}
}

#include "hip/hip_runtime.h"
#include "cusp/complex.h"
#define EPS 1e-50
#define complex_t cusp::complex<double>
__device__ double calc_F(const int i, const int j, const int k, double *d_data, 
						double *d_params, double *d_const, const int N)
{
%JAC1_FUNCTIONS%
}



/****************************************************************************

Optimized version of STM propagator using multiple sub-arcs and multiple arcs

*****************************************************************************/
#include <stdio.h>
extern "C" __global__ 
	void stt1_Fstar_kernel(double *d_x0, double *d_F, double *d_params, double *d_const, 
		double* d_phi0, double *d_k, double *d_k2, double *d_phi1, const double h,
		const double k_mul1, const double k_mul2, const int N, const int P, double *arcSequence, int x0Size, int part)
{
	double x0_[%NUMSTATES%];
	int j;
	int realcol, realrow;

	if(part == 1)
	{
		// Copy states for every sub-arc into shared memory for current sub-step
		// N values per thread, P threads
		if(blockIdx.x<%NUMSTATES% && blockIdx.y<%NUMSTATES% && threadIdx.x < P)
		{
			
			for(j=0;j<%NUMSTATES%;j++)
			{
				realcol = blockIdx.z*x0Size + j*P+ threadIdx.x;
				x0_[j] = d_x0[realcol];
			}
	
			realcol = P*blockIdx.y + threadIdx.x;
			d_F[blockIdx.z*N*N*P + %NUMSTATES%*P*blockIdx.x + realcol] = calc_F(blockIdx.x,blockIdx.y,arcSequence(blockIdx.z),x0_, d_params, d_const, %NUMSTATES%);
		}
	}
	else if(part == 2)
	{
		// Part 2
		double phidot;
		if(blockIdx.x<%NUMSTATES% && blockIdx.y<%NUMSTATES% && threadIdx.x < P)
		{
			// Dot product of F and phi
			phidot = 0;
			for(j=0;j<%NUMSTATES%;j++)
			{
				// realrow = j*P + threadIdx.x;
				realcol = j*P + threadIdx.x;
				phidot += d_F[blockIdx.z*N*N*P + %NUMSTATES%*P*blockIdx.x + realcol]*
							(d_phi0[blockIdx.z*N*N*P + blockIdx.y*%NUMSTATES%*P + realcol] + d_k[blockIdx.z*N*N*P + blockIdx.y*%NUMSTATES%*P + realcol]*k_mul2);
			}
			phidot = h*phidot;
			
			// Store into "k" matrix in col-major form  k_i = h*phidot
			realrow = blockIdx.x*P + threadIdx.x;
			
			d_k2[blockIdx.z*N*N*P + blockIdx.y*%NUMSTATES%*P + realrow] = phidot;			
			d_phi1[blockIdx.z*N*N*P + blockIdx.y*%NUMSTATES%*P + realrow] += k_mul1*phidot;
		}
	}
}
